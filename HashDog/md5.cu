#include "hip/hip_runtime.h"
/**
 **********************************************************************
 ** Copyright (C) 1990, RSA Data Security, Inc. All rights reserved. **
 **                                                                  **
 ** License to copy and use this software is granted provided that   **
 ** it is identified as the "RSA Data Security, Inc. MD5 Message     **
 ** Digest Algorithm" in all material mentioning or referencing this **
 ** software or this function.                                       **
 **                                                                  **
 ** License is also granted to make and use derivative works         **
 ** provided that such works are identified as "derived from the RSA **
 ** Data Security, Inc. MD5 Message Digest Algorithm" in all         **
 ** material mentioning or referencing the derived work.             **
 **                                                                  **
 ** RSA Data Security, Inc. makes no representations concerning      **
 ** either the merchantability of this software or the suitability   **
 ** of this software for any particular purpose.  It is provided "as **
 ** is" without express or implied warranty of any kind.             **
 **                                                                  **
 ** These notices must be retained in any copies of any part of this **
 ** documentation and/or software.                                   **
 **********************************************************************
 */

#include "md5.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__device__ void md5_hash(unsigned char* data, uint32_t length, uint32_t *a1, uint32_t *b1, uint32_t *c1, uint32_t *d1) {
	const uint32_t a0 = 0x67452301;
	const uint32_t b0 = 0xEFCDAB89;
	const uint32_t c0 = 0x98BADCFE;
	const uint32_t d0 = 0x10325476;

	uint32_t a = 0;
	uint32_t b = 0;
	uint32_t c = 0;
	uint32_t d = 0;

	uint32_t vals[14] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0 };

	int i = 0;
	for (i = 0; i < length; i++) {
		vals[i / 4] |= data[i] << ((i % 4) * 8);
	}

	vals[i / 4] |= 0x80 << ((i % 4) * 8);

	uint32_t bitlen = length * 8;

#define in0  (vals[0])//x
#define in1  (vals[1])//y
#define in2  (vals[2])//z
#define in3  (vals[3])
#define in4  (vals[4])
#define in5  (vals[5])
#define in6  (vals[6])
#define in7  (vals[7])
#define in8  (vals[8])
#define in9  (vals[9])
#define in10 (vals[10])
#define in11 (vals[11])
#define in12 (vals[12])
#define in13 (vals[13])
#define in14 (bitlen) //w = bit length
#define in15 (0)

	//Initialize hash value for this chunk:
	a = a0;
	b = b0;
	c = c0;
	d = d0;

	/* Round 1 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
	FF(a, b, c, d, in0, S11, 3614090360); /* 1 */
	FF(d, a, b, c, in1, S12, 3905402710); /* 2 */
	FF(c, d, a, b, in2, S13, 606105819); /* 3 */
	FF(b, c, d, a, in3, S14, 3250441966); /* 4 */
	FF(a, b, c, d, in4, S11, 4118548399); /* 5 */
	FF(d, a, b, c, in5, S12, 1200080426); /* 6 */
	FF(c, d, a, b, in6, S13, 2821735955); /* 7 */
	FF(b, c, d, a, in7, S14, 4249261313); /* 8 */
	FF(a, b, c, d, in8, S11, 1770035416); /* 9 */
	FF(d, a, b, c, in9, S12, 2336552879); /* 10 */
	FF(c, d, a, b, in10, S13, 4294925233); /* 11 */
	FF(b, c, d, a, in11, S14, 2304563134); /* 12 */
	FF(a, b, c, d, in12, S11, 1804603682); /* 13 */
	FF(d, a, b, c, in13, S12, 4254626195); /* 14 */
	FF(c, d, a, b, in14, S13, 2792965006); /* 15 */
	FF(b, c, d, a, in15, S14, 1236535329); /* 16 */

	/* Round 2 */
#define S21 5
#define S22 9
#define S23 14
#define S24 20
	GG(a, b, c, d, in1, S21, 4129170786); /* 17 */
	GG(d, a, b, c, in6, S22, 3225465664); /* 18 */
	GG(c, d, a, b, in11, S23, 643717713); /* 19 */
	GG(b, c, d, a, in0, S24, 3921069994); /* 20 */
	GG(a, b, c, d, in5, S21, 3593408605); /* 21 */
	GG(d, a, b, c, in10, S22, 38016083); /* 22 */
	GG(c, d, a, b, in15, S23, 3634488961); /* 23 */
	GG(b, c, d, a, in4, S24, 3889429448); /* 24 */
	GG(a, b, c, d, in9, S21, 568446438); /* 25 */
	GG(d, a, b, c, in14, S22, 3275163606); /* 26 */
	GG(c, d, a, b, in3, S23, 4107603335); /* 27 */
	GG(b, c, d, a, in8, S24, 1163531501); /* 28 */
	GG(a, b, c, d, in13, S21, 2850285829); /* 29 */
	GG(d, a, b, c, in2, S22, 4243563512); /* 30 */
	GG(c, d, a, b, in7, S23, 1735328473); /* 31 */
	GG(b, c, d, a, in12, S24, 2368359562); /* 32 */

	/* Round 3 */
#define S31 4
#define S32 11
#define S33 16
#define S34 23
	HH(a, b, c, d, in5, S31, 4294588738); /* 33 */
	HH(d, a, b, c, in8, S32, 2272392833); /* 34 */
	HH(c, d, a, b, in11, S33, 1839030562); /* 35 */
	HH(b, c, d, a, in14, S34, 4259657740); /* 36 */
	HH(a, b, c, d, in1, S31, 2763975236); /* 37 */
	HH(d, a, b, c, in4, S32, 1272893353); /* 38 */
	HH(c, d, a, b, in7, S33, 4139469664); /* 39 */
	HH(b, c, d, a, in10, S34, 3200236656); /* 40 */
	HH(a, b, c, d, in13, S31, 681279174); /* 41 */
	HH(d, a, b, c, in0, S32, 3936430074); /* 42 */
	HH(c, d, a, b, in3, S33, 3572445317); /* 43 */
	HH(b, c, d, a, in6, S34, 76029189); /* 44 */
	HH(a, b, c, d, in9, S31, 3654602809); /* 45 */
	HH(d, a, b, c, in12, S32, 3873151461); /* 46 */
	HH(c, d, a, b, in15, S33, 530742520); /* 47 */
	HH(b, c, d, a, in2, S34, 3299628645); /* 48 */

	/* Round 4 */
#define S41 6
#define S42 10
#define S43 15
#define S44 21
	II(a, b, c, d, in0, S41, 4096336452); /* 49 */
	II(d, a, b, c, in7, S42, 1126891415); /* 50 */
	II(c, d, a, b, in14, S43, 2878612391); /* 51 */
	II(b, c, d, a, in5, S44, 4237533241); /* 52 */
	II(a, b, c, d, in12, S41, 1700485571); /* 53 */
	II(d, a, b, c, in3, S42, 2399980690); /* 54 */
	II(c, d, a, b, in10, S43, 4293915773); /* 55 */
	II(b, c, d, a, in1, S44, 2240044497); /* 56 */
	II(a, b, c, d, in8, S41, 1873313359); /* 57 */
	II(d, a, b, c, in15, S42, 4264355552); /* 58 */
	II(c, d, a, b, in6, S43, 2734768916); /* 59 */
	II(b, c, d, a, in13, S44, 1309151649); /* 60 */
	II(a, b, c, d, in4, S41, 4149444226); /* 61 */
	II(d, a, b, c, in11, S42, 3174756917); /* 62 */
	II(c, d, a, b, in2, S43, 718787259); /* 63 */
	II(b, c, d, a, in9, S44, 3951481745); /* 64 */

	a += a0;
	b += b0;
	c += c0;
	d += d0;

	*a1 = a;
	*b1 = b;
	*c1 = c;
	*d1 = d;
}


__device__ __host__ bool next(uint8_t* length, char* word, uint32_t increment) {
	uint32_t idx = 0;
	uint32_t add = 0;

	while (increment > 0 && idx < CONST_WORD_LIMIT) {
		if (idx >= *length && increment > 0) {
			increment--;
		}

		add = increment + word[idx];
		word[idx] = add % CONST_CHARSET_LENGTH;
		increment = add / CONST_CHARSET_LENGTH;
		idx++;
	}

	if (idx > *length) {
		*length = idx;
	}

	if (idx > CONST_WORD_LIMIT) {
		return false;
	}

	return true;
}

__global__ void md5_crack(uint8_t* wordLength, char* charsetWord, uint32_t* searched_hash, char* word_cracked) {
	uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;
	__shared__ char sharedCharset[CONST_CHARSET_LIMIT];

	/* Thread variables */
	char threadCharsetWord[CONST_WORD_LIMIT];
	char threadTextWord[CONST_WORD_LIMIT];
	uint8_t threadWordLength = *wordLength;
	uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

	memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
	memcpy(sharedCharset, d_abedeca, sizeof(uint8_t) * CONST_CHARSET_LIMIT);

	/* Increment current word by thread index */
	next(&threadWordLength, threadCharsetWord, idx);

	for (uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++) {
		for (uint32_t i = 0; i < threadWordLength; i++) {
			threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
		}



		md5_hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

		if (threadHash01 == searched_hash[0] && threadHash02 == searched_hash[1] && threadHash03 == searched_hash[2] && threadHash04 == searched_hash[3]) {

			memcpy(word_cracked, threadTextWord, threadWordLength);
		}
		if (!next(&threadWordLength, threadCharsetWord, 1)) {
			break;
		}
	}
}

void crack_md5(char* input, int pass_length) {
	char* word;
	int CONST_PASSWORD_LENGTH = pass_length;
	hipMallocManaged(&word, CONST_PASSWORD_LENGTH);
	char abeceda[CONST_CHARSET_LIMIT];
	char* heslo = (char*)malloc(CONST_PASSWORD_LENGTH);

	uint8_t* wordLength;
	hipMallocManaged(&wordLength, sizeof(uint8_t));



	unsigned char* hex = new unsigned char[256];
	memcpy(hex, input, 32 + 1);

	uint32_t* md5Hash;
	hipMallocManaged(&md5Hash, 4 * sizeof(uint32_t));
	for (int i = 0; i < 4; i++)
	{
		char tmp[16];

		memcpy(tmp, hex + i*8, 8);
		sscanf(tmp, "%x", &md5Hash[i]);
		md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
	}



	printf("hladany hash: %s", hex);


	memset(word, 0, CONST_PASSWORD_LENGTH);
	memset(heslo, 0, CONST_PASSWORD_LENGTH);
	memcpy(abeceda, CONST_CHARSET, CONST_CHARSET_LENGTH);
	*wordLength = CONST_PASSWORD_LENGTH;

	printf("\nword length: %u", *wordLength);


	hipSetDevice(0);



	char* d_word, *d_word_cracked;
	hipMallocManaged((void**)&d_word, sizeof(uint8_t) * CONST_PASSWORD_LENGTH);
	hipMallocManaged((void**)&d_word_cracked, sizeof(uint8_t) * CONST_PASSWORD_LENGTH);



	hipMemcpyToSymbol(HIP_SYMBOL(d_abedeca), abeceda, sizeof(uint8_t) * CONST_CHARSET_LIMIT, 0, hipMemcpyHostToDevice);


	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
	while (true) {

		bool result = false;
		bool found = false;


		
		(md5_crack<<<TOTAL_BLOCKS, TOTAL_THREADS>>> (wordLength, d_word, md5Hash, d_word_cracked));
		hipDeviceSynchronize();
		
		result = next(wordLength, d_word, TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS);


		
		if ((*d_word_cracked != NULL) && (d_word_cracked[0] != '\0')) {
			std::cout << "\nCracked " << d_word_cracked << std::endl;
			break;
		}


		if (!result || found) {
			if (!result && !found) {
				std::cout << "\nFound nothing (host)" << std::endl;
			}

			break;
		}
	}
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();




	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
	std::cout << "Hash computation time: " << duration << "us";



	hipFree(word);
}

char *bin2hex(const unsigned char *bin, size_t len)
{
	char   *out;
	size_t  i;

	if (bin == NULL || len == 0)
		return NULL;

	out = (char *) malloc(len * 2 + 1);
	for (i = 0; i < len; i++) {
		out[i * 2] = "0123456789ABCDEF"[bin[i] >> 4];
		out[i * 2 + 1] = "0123456789ABCDEF"[bin[i] & 0x0F];
	}
	out[len * 2] = '\0';

	return out;
}
