#include "hip/hip_runtime.h"
/**
 **********************************************************************
 ** Copyright (C) 1990, RSA Data Security, Inc. All rights reserved. **
 **                                                                  **
 ** License to copy and use this software is granted provided that   **
 ** it is identified as the "RSA Data Security, Inc. MD5 Message     **
 ** Digest Algorithm" in all material mentioning or referencing this **
 ** software or this function.                                       **
 **                                                                  **
 ** License is also granted to make and use derivative works         **
 ** provided that such works are identified as "derived from the RSA **
 ** Data Security, Inc. MD5 Message Digest Algorithm" in all         **
 ** material mentioning or referencing the derived work.             **
 **                                                                  **
 ** RSA Data Security, Inc. makes no representations concerning      **
 ** either the merchantability of this software or the suitability   **
 ** of this software for any particular purpose.  It is provided "as **
 ** is" without express or implied warranty of any kind.             **
 **                                                                  **
 ** These notices must be retained in any copies of any part of this **
 ** documentation and/or software.                                   **
 **********************************************************************
 */

#include "md5.h"

char g_cracked[CONST_WORD_LIMIT];

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


__device__ void md5_hash(unsigned char* data, uint32_t length, uint32_t *a1, uint32_t *b1, uint32_t *c1, uint32_t *d1) {
	const uint32_t a0 = 0x67452301;
	const uint32_t b0 = 0xEFCDAB89;
	const uint32_t c0 = 0x98BADCFE;
	const uint32_t d0 = 0x10325476;

	uint32_t a = 0;
	uint32_t b = 0;
	uint32_t c = 0;
	uint32_t d = 0;

	uint32_t vals[14] = { 0,0,0,0,0,0,0,0,0,0,0,0,0,0 };

	int i = 0;
	for (i = 0; i < length; i++) {
		vals[i / 4] |= data[i] << ((i % 4) * 8);
	}

	vals[i / 4] |= 0x80 << ((i % 4) * 8);

	uint32_t bitlen = length * 8;

#define in0  (vals[0])//x
#define in1  (vals[1])//y
#define in2  (vals[2])//z
#define in3  (vals[3])
#define in4  (vals[4])
#define in5  (vals[5])
#define in6  (vals[6])
#define in7  (vals[7])
#define in8  (vals[8])
#define in9  (vals[9])
#define in10 (vals[10])
#define in11 (vals[11])
#define in12 (vals[12])
#define in13 (vals[13])
#define in14 (bitlen) //w = bit length
#define in15 (0)

	//Initialize hash value for this chunk:
	a = a0;
	b = b0;
	c = c0;
	d = d0;

	/* Round 1 */
#define S11 7
#define S12 12
#define S13 17
#define S14 22
	FF(a, b, c, d, in0, S11, 3614090360); /* 1 */
	FF(d, a, b, c, in1, S12, 3905402710); /* 2 */
	FF(c, d, a, b, in2, S13, 606105819); /* 3 */
	FF(b, c, d, a, in3, S14, 3250441966); /* 4 */
	FF(a, b, c, d, in4, S11, 4118548399); /* 5 */
	FF(d, a, b, c, in5, S12, 1200080426); /* 6 */
	FF(c, d, a, b, in6, S13, 2821735955); /* 7 */
	FF(b, c, d, a, in7, S14, 4249261313); /* 8 */
	FF(a, b, c, d, in8, S11, 1770035416); /* 9 */
	FF(d, a, b, c, in9, S12, 2336552879); /* 10 */
	FF(c, d, a, b, in10, S13, 4294925233); /* 11 */
	FF(b, c, d, a, in11, S14, 2304563134); /* 12 */
	FF(a, b, c, d, in12, S11, 1804603682); /* 13 */
	FF(d, a, b, c, in13, S12, 4254626195); /* 14 */
	FF(c, d, a, b, in14, S13, 2792965006); /* 15 */
	FF(b, c, d, a, in15, S14, 1236535329); /* 16 */

	/* Round 2 */
#define S21 5
#define S22 9
#define S23 14
#define S24 20
	GG(a, b, c, d, in1, S21, 4129170786); /* 17 */
	GG(d, a, b, c, in6, S22, 3225465664); /* 18 */
	GG(c, d, a, b, in11, S23, 643717713); /* 19 */
	GG(b, c, d, a, in0, S24, 3921069994); /* 20 */
	GG(a, b, c, d, in5, S21, 3593408605); /* 21 */
	GG(d, a, b, c, in10, S22, 38016083); /* 22 */
	GG(c, d, a, b, in15, S23, 3634488961); /* 23 */
	GG(b, c, d, a, in4, S24, 3889429448); /* 24 */
	GG(a, b, c, d, in9, S21, 568446438); /* 25 */
	GG(d, a, b, c, in14, S22, 3275163606); /* 26 */
	GG(c, d, a, b, in3, S23, 4107603335); /* 27 */
	GG(b, c, d, a, in8, S24, 1163531501); /* 28 */
	GG(a, b, c, d, in13, S21, 2850285829); /* 29 */
	GG(d, a, b, c, in2, S22, 4243563512); /* 30 */
	GG(c, d, a, b, in7, S23, 1735328473); /* 31 */
	GG(b, c, d, a, in12, S24, 2368359562); /* 32 */

	/* Round 3 */
#define S31 4
#define S32 11
#define S33 16
#define S34 23
	HH(a, b, c, d, in5, S31, 4294588738); /* 33 */
	HH(d, a, b, c, in8, S32, 2272392833); /* 34 */
	HH(c, d, a, b, in11, S33, 1839030562); /* 35 */
	HH(b, c, d, a, in14, S34, 4259657740); /* 36 */
	HH(a, b, c, d, in1, S31, 2763975236); /* 37 */
	HH(d, a, b, c, in4, S32, 1272893353); /* 38 */
	HH(c, d, a, b, in7, S33, 4139469664); /* 39 */
	HH(b, c, d, a, in10, S34, 3200236656); /* 40 */
	HH(a, b, c, d, in13, S31, 681279174); /* 41 */
	HH(d, a, b, c, in0, S32, 3936430074); /* 42 */
	HH(c, d, a, b, in3, S33, 3572445317); /* 43 */
	HH(b, c, d, a, in6, S34, 76029189); /* 44 */
	HH(a, b, c, d, in9, S31, 3654602809); /* 45 */
	HH(d, a, b, c, in12, S32, 3873151461); /* 46 */
	HH(c, d, a, b, in15, S33, 530742520); /* 47 */
	HH(b, c, d, a, in2, S34, 3299628645); /* 48 */

	/* Round 4 */
#define S41 6
#define S42 10
#define S43 15
#define S44 21
	II(a, b, c, d, in0, S41, 4096336452); /* 49 */
	II(d, a, b, c, in7, S42, 1126891415); /* 50 */
	II(c, d, a, b, in14, S43, 2878612391); /* 51 */
	II(b, c, d, a, in5, S44, 4237533241); /* 52 */
	II(a, b, c, d, in12, S41, 1700485571); /* 53 */
	II(d, a, b, c, in3, S42, 2399980690); /* 54 */
	II(c, d, a, b, in10, S43, 4293915773); /* 55 */
	II(b, c, d, a, in1, S44, 2240044497); /* 56 */
	II(a, b, c, d, in8, S41, 1873313359); /* 57 */
	II(d, a, b, c, in15, S42, 4264355552); /* 58 */
	II(c, d, a, b, in6, S43, 2734768916); /* 59 */
	II(b, c, d, a, in13, S44, 1309151649); /* 60 */
	II(a, b, c, d, in4, S41, 4149444226); /* 61 */
	II(d, a, b, c, in11, S42, 3174756917); /* 62 */
	II(c, d, a, b, in2, S43, 718787259); /* 63 */
	II(b, c, d, a, in9, S44, 3951481745); /* 64 */

	a += a0;
	b += b0;
	c += c0;
	d += d0;

	*a1 = a;
	*b1 = b;
	*c1 = c;
	*d1 = d;
}


__device__ __host__ bool next(uint8_t* length, char* word, uint32_t increment) {
	uint32_t idx = 0;
	uint32_t add = 0;

	while (increment > 0 && idx < CONST_WORD_LIMIT) {
		if (idx >= *length && increment > 0) {
			increment--;
		}

		add = increment + word[idx];
		word[idx] = add % CONST_CHARSET_LENGTH;
		increment = add / CONST_CHARSET_LENGTH;
		idx++;
	}

	if (idx > *length) {
		*length = idx;
	}

	if (idx > CONST_WORD_LENGTH_MAX) {
		return false;
	}

	return true;
}

__global__ void md5_crack(uint8_t* wordLength, char* charsetWord, uint32_t* searched_hash, char* g_device_cracked) {
	uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;
	/* Shared variables */
	__shared__ char sharedCharset[CONST_CHARSET_LIMIT];

	/* Thread variables */
	char threadCharsetWord[CONST_WORD_LIMIT];
	char threadTextWord[CONST_WORD_LIMIT];
	uint8_t threadWordLength = *wordLength;
	//printf("%u", *wordLength);
	uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

	/* Copy everything to local memory */
	memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
	//memcpy(&threadWordLength, wordLength, sizeof(uint8_t));
	memcpy(sharedCharset, g_deviceCharset, sizeof(uint8_t) * CONST_CHARSET_LIMIT);

	/* Increment current word by thread index */
	next(&threadWordLength, threadCharsetWord, idx);

	for (uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++) {
		for (uint32_t i = 0; i < threadWordLength; i++) {
			threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
		}

		//if (threadWordLength == 4)
		//{
		//	(threadTextWord[3] << 24) | (threadTextWord[2] << 16) | (threadTextWord[1] << 8) | (threadTextWord[0]);
		//	if (threadTextWord[0] == 'f' && threadTextWord[0] == 'e' && threadTextWord[0] == 'r' && threadTextWord[0] == 'o')
		//	{
		//		printf("mam ho");
		//	}

		//}

		md5_hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

		if (threadHash01 == searched_hash[0] && threadHash02 == searched_hash[1] && threadHash03 == searched_hash[2] && threadHash04 == searched_hash[3]) {
			printf("%s", threadTextWord);

			memcpy(g_device_cracked, threadTextWord, threadWordLength);
		}

		if (!next(&threadWordLength, threadCharsetWord, 1)) {
			break;
		}
	}
}

void crack_md5() {
	const short PASSWORD_LENGTH = 4;
	const int HASH_SIZE = 32;
	char* g_word;
	hipMallocManaged(&g_word, CONST_WORD_LIMIT);
	char g_charset[CONST_CHARSET_LIMIT];
	char g_cracked[CONST_WORD_LIMIT];
	uint8_t* g_wordLength;
	hipMallocManaged(&g_wordLength, sizeof(uint8_t));


	char* searched_string = new char[PASSWORD_LENGTH + 1];
	unsigned char* searched_digest = new unsigned char[HASH_SIZE];
	unsigned char* hex = new unsigned char[256];
	memcpy(searched_string, "fero", PASSWORD_LENGTH + 1);
	//memcpy(hex, "ac6545028a5d090df842d8d9d674fc6e", 32 + 1);
	memcpy(hex, "d3c1a5b4d60ae870a12fbde520ea11f9", 32 + 1);
	//Md5Hash *md5_hash = new Md5Hash();
	//md5_hash->hash_message(searched_string, searched_digest);
	//printf("%s", searched_digest);


	

	//uint32_t md5Hash[4];
	uint32_t* md5Hash;
	hipMallocManaged(&md5Hash, 4 * sizeof(uint32_t));
	for (int i = 0; i < 4; i++)
	{
		char tmp[16];

	//AC6545028A5D090DF842D8D9D674FC6E

		memcpy(tmp, hex + i*8, 8);
		sscanf(tmp, "%x", &md5Hash[i]);
		md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
	}


	printf("lol");

	printf("hladany hash: %u %u %u %u", md5Hash[0], md5Hash[1], md5Hash[2], md5Hash[3]);


	memset(g_word, 0, CONST_WORD_LIMIT);
	memset(g_cracked, 0, CONST_WORD_LIMIT);
	memcpy(g_charset, CONST_CHARSET, CONST_CHARSET_LENGTH);
	*g_wordLength = CONST_WORD_LENGTH_MIN;

	printf("\n word length: %u", *g_wordLength);


	hipSetDevice(0);

	/* Time */
	//hipEvent_t clockBegin;
	//hipEvent_t clockLast;


	//hipEventCreate(&clockBegin);
	//hipEventCreate(&clockLast);
	//hipEventRecord(clockBegin, 0);

	/*char* words;
	hipMallocManaged((void**)&words, sizeof(uint8_t) * CONST_WORD_LIMIT);*/

	char* g_deviceCracked;
	hipMallocManaged((void**)&g_deviceCracked, sizeof(uint8_t) * CONST_WORD_LIMIT);
	//hipMallocManaged((void**)&g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT);



	hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(uint8_t) * CONST_CHARSET_LIMIT, 0, hipMemcpyHostToDevice);
	//hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyHostToDevice);

	/* Allocate on each device */

	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	//hipEventRecord(start);

	std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
	while (true) {

		bool result = false;
		bool found = false;


		/* Copy current data */
		//memcpy(words, g_word, sizeof(uint8_t) * CONST_WORD_LIMIT);
		//int n_blocks = CONST_CHARSET_LENGTH

		/* Start kernel */
		(md5_crack<<<TOTAL_BLOCKS, TOTAL_THREADS>>> (g_wordLength, g_word, md5Hash, g_deviceCracked));
		hipDeviceSynchronize();
		//printf("%s", hipGetErrorString(hipGetLastError()));


		//printf("%u\n", g_wordLength);
		/* Global increment */
		result = next(g_wordLength, g_word, TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS);


		/* Display progress */
		//char word[CONST_WORD_LIMIT];

		//for (int i = 0; i < *g_wordLength; i++) {
		//	word[i] = g_charset[g_word[i]];
		//}



		//std::cout << "Notice: currently at " << std::string(word, g_wordLength) << " (" << (uint32_t)g_wordLength << ")" << std::endl;


		/* Synchronize now */

		/* Check result */
		if ((*g_deviceCracked != NULL) && (g_deviceCracked[0] != '\0')) {
			std::cout << "Notice: cracked " << g_deviceCracked << std::endl;
			break;
		}


		if (!result || found) {
			if (!result && !found) {
				std::cout << "Notice: found nothing (host)" << std::endl;
			}

			break;
		}
	}
	std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();


	//hipEventRecord(stop);

	//float milliseconds = 0;
	//hipEventElapsedTime(&milliseconds, start, stop);

	//hipEventDestroy(start);
	//hipEventDestroy(stop);

	//std::cout << "Time: " << milliseconds << " ms";


	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
	std::cout << "Hash computation time: " << duration << "us";



	/* Free on each device */
	hipFree(g_word);
}

char *bin2hex(const unsigned char *bin, size_t len)
{
	char   *out;
	size_t  i;

	if (bin == NULL || len == 0)
		return NULL;

	out = (char *) malloc(len * 2 + 1);
	for (i = 0; i < len; i++) {
		out[i * 2] = "0123456789ABCDEF"[bin[i] >> 4];
		out[i * 2 + 1] = "0123456789ABCDEF"[bin[i] & 0x0F];
	}
	out[len * 2] = '\0';

	return out;
}
